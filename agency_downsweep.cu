#include "hip/hip_runtime.h"
#include <numeric>
#include <iostream>

#include <moderngpu/transform.hxx>   // for cta_launch.
#include <moderngpu/memory.hxx>      // for mem_t.
#include <moderngpu/kernel_scan.hxx>
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <agency/experimental.hpp>
#include "measure_bandwidth_of_invocation.hpp"
#include "algorithm/copy.hpp"
#include "algorithm/for_loop.hpp"
#include "algorithm/transform.hpp"
#include "algorithm/inclusive_scan.hpp"
#include "collective_scanner.hpp"


template<size_t block_size, size_t grain_size = 1>
auto static_grid(int num_blocks) ->
  decltype(agency::cuda::par(num_blocks, agency::cuda::experimental::static_concurrent_execution_policy<block_size, grain_size>()))
{
  return agency::cuda::par(num_blocks, agency::cuda::experimental::static_concurrent_execution_policy<block_size, grain_size>());
}

template<size_t block_size, size_t grain_size = 1>
using static_grid_agent = agency::parallel_group<agency::experimental::static_concurrent_agent<block_size, grain_size>>;

using namespace mgpu;
using namespace agency::experimental;


// XXX this costs a ton of registers
template<bool exclusive, size_t group_size, size_t grain_size, class Range1, class Range2, class T, class BinaryOperation>
__device__
T collective_scan_with_carry(agency::experimental::static_concurrent_agent<group_size,grain_size>& self, const Range1& input, Range2&& output, T carry_in, BinaryOperation binary_op)
{
  using namespace agency::experimental;

  constexpr int tile_size = group_size * grain_size;

  static_assert(!exclusive, "exclusive scan unimplemented!");

  __shared__ union
  {
    collective_scanner<T,group_size> scanner;
    array<T,tile_size> tile_storage;
  } shared;

  for(int offset = 0; offset < input.size(); offset += tile_size)
  {
    auto partial_tile_size = min(tile_size, int(input.size() - offset));
    auto input_tile = counted(input, offset, bounded_int<tile_size>(partial_tile_size));

    // create a view of shared memory with as many elements as the input tile
    auto view_of_shared_tile = counted(shared.tile_storage, input_tile.size());

    // copy the input tile into shared memory
    collective_copy(self, input_tile, view_of_shared_tile);

    // tile shared memory into subtiles of size grain_size
    auto shared_subtiles = chunk(view_of_shared_tile, grain_size);
    auto view_of_shared_subtile = shared_subtiles.chunk_or_empty(self.rank());
    int num_subtiles = shared_subtiles.size();

    // copy the subtile from shared memory into registers
    short_vector<T,grain_size> local_subtile = view_of_shared_subtile;

    // each agent does an in-place inclusive scan of its local array
    ::inclusive_scan(local_subtile, local_subtile, binary_op);

    // each thread contributes a summand for a group-wide exclusive scan 
    optional<T> summand = local_subtile.back_or_none();

    // wait until all agents have their summand before using the scanner
    self.wait();

    // collectively compute the exclusive scan of the summands in-place
    carry_in = shared.scanner.inplace_exclusive_scan(self, summand, num_subtiles, carry_in, binary_op);

    // to produce the final inclusive scan, add the thread's carry-in to the scan of its local subtile
    // store each thread's result directly to the tile in shared memory
    transform(local_subtile, view_of_shared_subtile, [=](const T& element)
    {
      return binary_op(*summand, element);
    });

    // wait for each thread to complete their result
    self.wait();

    // copy the tile from shared memory into the output tile
    auto output_tile = counted(output, offset, bounded_int<tile_size>(partial_tile_size));
    collective_copy(self, view_of_shared_tile, output_tile);
  }

  return carry_in;
}


// XXX abstracting this into a function introduces a register
template<bool exclusive, size_t group_size, size_t grain_size, class Range1, class Range2, class T, class BinaryOperation>
__device__
T bounded_scan_tile(agency::experimental::static_concurrent_agent<group_size,grain_size>& self, const Range1& input_tile, Range2&& output_tile, T carry_in, BinaryOperation binary_op)
{
  using namespace agency::experimental;

  static_assert(!exclusive, "exclusive scan unimplemented!");

  __shared__ union
  {
    collective_scanner<T,group_size> scanner;
    array<T,group_size * grain_size> tile_storage;
  } shared;

  // create a view of shared memory with as many elements as the input tile
  auto view_of_shared_tile = counted(shared.tile_storage, input_tile.size());

  // copy the input tile into shared memory
  collective_copy(self, input_tile, view_of_shared_tile);

  // tile shared memory into subtiles of size grain_size
  auto shared_subtiles = chunk(view_of_shared_tile, grain_size);
  auto view_of_shared_subtile = shared_subtiles.chunk_or_empty(self.rank());
  int num_subtiles = shared_subtiles.size();

  // copy the subtile from shared memory into registers
  // XXX should call something like to_container() here
  //     statically-sized ranges would be copied into an array<T,N>
  //     bounded-sized ranges would be copied into a short_vector<T,N>
  short_vector<T,grain_size> local_subtile = view_of_shared_subtile;

  // each agent does an in-place inclusive scan of its local array
  ::inclusive_scan(local_subtile, local_subtile, binary_op);

  // each thread contributes a summand for a group-wide exclusive scan 
  optional<T> summand = local_subtile.back_or_none();

  // wait until all agents have their summand before using the scanner
  self.wait();

  // collectively compute the exclusive scan of the summands in-place
  T carry_out = shared.scanner.inplace_exclusive_scan(self, summand, num_subtiles, carry_in, binary_op);

  // to produce the final inclusive scan, add the thread's carry-in to the scan of its local subtile
  // store each thread's result directly to the tile in shared memory
  // XXX for some reason capturing by reference significantly inflates the register requirements
  //     so capture summand by value
  transform(local_subtile, view_of_shared_subtile, [=](const T& element)
  {
    return binary_op(*summand, element);
  });

  // wait for each thread to complete their result
  self.wait();

  // copy the tile from shared memory into the output tile
  collective_copy(self, view_of_shared_tile, output_tile);

  return carry_out;
}


template<size_t group_size, size_t grain_size, typename input_it1, typename input_it2, typename output_it, typename op_t>
void inclusive_scan_tiles(input_it1 input, int count, input_it2 inits, output_it output, op_t binary_op, context_t& context)
{
  constexpr int tile_size = group_size * grain_size;

  // embed tile_size into the type system
  constexpr auto bounded_tile_size = bounded_int<tile_size>(tile_size);

  typedef typename std::iterator_traits<input_it1>::value_type T;

  auto input_view  = span<T>(input, count);
  auto output_view = span<T>(output, count);

  auto input_tiles = chunk(input_view, bounded_tile_size);
  auto output_tiles = chunk(output_view, bounded_tile_size);

  auto downsweep_k = [=] __device__ (static_grid_agent<group_size,grain_size>& self)
  {
    int cta = self.outer().index();

    auto input_tile = input_tiles[cta]; 
    auto output_tile = output_tiles[cta];

    bounded_scan_tile<false>(self.inner(), input_tile, output_tile, inits[cta], binary_op); 
    //collective_scan_with_carry<false>(self.inner(), input_tile, output_tile, inits[cta], binary_op);
  };
  agency::bulk_invoke(static_grid<group_size,grain_size>(input_tiles.size()), downsweep_k);
}


int main()
{
  standard_context_t context;

  constexpr size_t group_size = 128;
  constexpr size_t grain_size = 11;
  constexpr size_t tile_size = group_size * grain_size;

  for(int i = 0; i < 30; ++i)
  {
    size_t n = 1 << i;

    std::cout << "testing n: " << n << std::endl;

    size_t num_tiles = (n + tile_size - 1) / tile_size;

    std::vector<int> input_host(n);
    std::default_random_engine rng(n);
    std::generate(input_host.begin(), input_host.end(), rng);

    std::vector<int> inits_host(num_tiles);
    std::generate(inits_host.begin(), inits_host.end(), rng);

    // Copy the data to the GPU.
    mem_t<int> input_device = to_mem(input_host, context);
    mem_t<int> inits_device = to_mem(inits_host, context);

    mem_t<int> output_device(n, context);
    inclusive_scan_tiles<group_size, grain_size>(input_device.data(), input_device.size(), inits_device.data(), output_device.data(), plus_t<int>(), context);

    // Get the result.
    std::vector<int> output_host = from_mem(output_device);

    // compare to reference
    std::vector<int> reference(n);

    auto input_tiles = agency::experimental::chunk(input_host, tile_size);
    auto output_tiles = agency::experimental::chunk(reference, tile_size);

    for(int i = 0; i < input_tiles.size(); ++i)
    {
      std::partial_sum(input_tiles[i].begin(), input_tiles[i].end(), output_tiles[i].begin());
      std::for_each(output_tiles[i].begin(), output_tiles[i].end(), [&](int &x)
      {
        x += inits_host[i];
      });
    }

    assert(reference == output_host);
  }

  size_t n = 1 << 30;
  size_t num_tiles = (n + tile_size - 1) / tile_size;

  std::vector<int> input_host(n);
  std::default_random_engine rng(n);
  std::generate(input_host.begin(), input_host.end(), rng);

  std::vector<int> inits_host(num_tiles);
  std::generate(inits_host.begin(), inits_host.end(), rng);

  // Copy the data to the GPU.
  mem_t<int> input_device = to_mem(input_host, context);
  mem_t<int> inits_device = to_mem(inits_host, context);

  mem_t<int> output_device(n, context);

  auto bandwidth = measure_bandwidth_of_invocation_in_gigabytes_per_second(100, 2 * sizeof(int) * n, [&]
  {
    inclusive_scan_tiles<group_size,grain_size>(input_device.data(), input_device.size(), inits_device.data(), output_device.data(), plus_t<int>(), context);
  });

  std::cout << "Mean bandwidth: " << bandwidth << " GB/s " << std::endl;

  std::cout << "OK" << std::endl;

  return 0;
}

